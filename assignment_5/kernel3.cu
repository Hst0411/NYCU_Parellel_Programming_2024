#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int* DeviceMemory, float stepX, float stepY, float lowerX, float lowerY, int maxIterations, int pitch, int group_size) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * group_size;
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * group_size;
    
    for(int k = 0; k < group_size; k++){
        for(int j = 0; j < group_size; j++){
            float c_re = lowerX + (thisX + j) * stepX;
            float c_im = lowerY + (thisY + k) * stepY;
            float z_re = c_re, z_im = c_im;

            int i;
            for (i = 0; i < maxIterations; ++i){

                if (z_re * z_re + z_im * z_im > 4.f) break;

                float new_re = z_re * z_re - z_im * z_im;
                float new_im = 2.f * z_re * z_im;
                z_re = c_re + new_re;
                z_im = c_im + new_im;
            }
            int* row = (int*)((char*)DeviceMemory + (thisY + k) * pitch);
            row[thisX + j] = i;
        }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int size = resX * resY * sizeof(int);
    size_t pitch = 0;
    int group_size = 4;

    int *HostMemory;
    hipHostAlloc(&HostMemory, size, hipHostMallocMapped);
    int *DeviceMemory;
    hipMallocPitch(&DeviceMemory, &pitch, resX * sizeof(int), resY);

    dim3 threadsPerBlock(25, 25);
    dim3 numBlocks(resX / threadsPerBlock.x / group_size, resY / threadsPerBlock.y / group_size);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(DeviceMemory, stepX, stepY, lowerX, lowerY, maxIterations, pitch, group_size);

    hipMemcpy2D(HostMemory, resX * sizeof(int), DeviceMemory, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, HostMemory, size);

    hipFree(DeviceMemory);
    hipHostFree(HostMemory);
}
